#include <hip/hip_runtime.h>
#include ""
#include "gpu_solver.h"

//typedef float T;

void GPU_DiffusionSolver<float>::callGPU(){

}
void GPU_DiffusionSolver<float>::copyH2D(){
	hipMalloc(&d_X, sizeof(float)*mesh.cellNum);
	hipMemcpy(d_X, X, sizeof(float)*mesh.cellNum, hipMemcpyHostToDevice);

	hipMalloc(&d_A, sizeof(float)*mesh.JA.size());
	hipMemcpy(d_A, A, sizeof(float)*mesh.JA.size(), hipMemcpyHostToDevice);
	hipMalloc(&d_JA, sizeof(float)*mesh.JA.size());
	hipMemcpy(d_JA, JA, sizeof(float)*mesh.JA.size(), hipMemcpyHostToDevice);
	hipMalloc(&d_IA, sizeof(float)*mesh.IA.size());
	hipMemcpy(d_IA, IA, sizeof(float)*mesh.IA.size(), hipMemcpyHostToDevice);
}
void GPU_DiffusionSolver<float>::copyD2H(){
	hipMemcpy(X, d_X, sizeof(float)*mesh.cellNum, hipMemcpyDeviceToHost);
}