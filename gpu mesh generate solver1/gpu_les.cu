#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include ""
#include "gpu_les.h"
#include <iostream>
/*
typedef float T;

void GPU_GS_Solver<T>::solve(T * A, T * IA, T * JA, T * X) {

}
*/

__global__ void add_kernel(float * a, float * b, float* c, int N){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < N){
		c[tid] = a[tid] + b[tid];
	}
	
}


__global__ void test(){
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	printf("I'm in kernel");
	printf("tid=%d",tid);
}
void GPU_GS_Solver<float>::solve(float * A, float * IA, float * JA, float * X) {
	test << <1, 64 >> >();
	
	std::cout << "hi" << std::endl;
}
