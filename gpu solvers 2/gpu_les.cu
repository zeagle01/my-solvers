#include "hip/hip_runtime.h"


#include "case.h"
#include "Solver.h"
#include "configReader.h"
#include "linearEquationSolver.h"
#include <hip/hip_runtime.h>
#include ""


/*
__global__ void kernel(CSR* eq, CellField* phi, Mesh* mesh){
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < mesh->cellNum){
		printf("numof%d", mesh->X[tid * 2+0]);
		printf("hello??");
	}
}
*/
/*
for (int c = 0; c < mesh.cellNum; c++) {
	double temp = 0;
	for (int i = mesh.IA[c] + 1; i < mesh.IA[c + 1]; i++) {
		temp += eq.A[i] * phi.inner[mesh.JA[i]];
	}
	phi.inner[c] = (eq.b[c] - temp) / eq.A[mesh.IA[c]];
}
*/
__global__ void kernel(double*A, double* b,int* IA, int *JA, double* X, double* preX,int N){
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < N){
			double temp = 0;
			for (int i = IA[tid] + 1; i < IA[tid + 1]; i++){
				temp += A[i] * preX[JA[i]];
			}
			X[tid] = (b[tid] - temp) / A[IA[tid]];
	}
}

void GPU_Jacobi_Solver::solve(CSR eq, CellField& phi, Mesh mesh){

	/*
	CSR* d_eq;
	CellField* d_phi;
	Mesh* d_mesh;
	hipMalloc(&d_eq, sizeof(CSR));
	hipMalloc(&d_phi, sizeof(CellField));
	hipMalloc(&d_mesh, sizeof(Mesh));
	hipMemcpy(d_mesh, &mesh, sizeof(Mesh), hipMemcpyHostToDevice);
	*/
	/*
	double *A =new double[eq.A.size()];
	for (int i = 0; i < eq.A.size(); i++){
		A[i] = eq.A[i];
	}
	*/
	double *A = eq.A.data();
	double *b = eq.b.data();
	int *IA = mesh.IA.data();
	int  *JA = mesh.JA.data();
	double* X = phi.inner.data();
	int N = mesh.cellNum;
	double*d_A,*d_b, *d_X, *d_preX;
	int*d_IA, *d_JA;


	
	hipMalloc(&d_X, sizeof(double)*phi.inner.size());
	hipMalloc(&d_preX, sizeof(double)*phi.inner.size());
	hipMalloc(&d_A, sizeof(double)*eq.A.size());
	hipMalloc(&d_b, sizeof(double)*eq.b.size());
	hipMalloc(&d_IA, sizeof(int)*mesh.IA.size());
	hipMalloc(&d_JA, sizeof(int)*mesh.JA.size());

	hipMemcpy(d_A, A, sizeof(double)*eq.A.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(double)*eq.b.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_X, X, sizeof(double)*phi.inner.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_preX, X, sizeof(double)*phi.inner.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_IA, IA, sizeof(int)*mesh.IA.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_JA, JA, sizeof(int)*mesh.JA.size(), hipMemcpyHostToDevice);

	int tpb = 256;
	int bpg =(  N+ 256 - 1) / tpb;

	double error = 0; int step = 0;
	do{
		CellField pre = phi;

		for (int it = 0; it < check_step; it++){
			hipMemcpy(d_preX, d_X, sizeof(double)*phi.inner.size(), hipMemcpyDeviceToDevice);
			kernel << <bpg, tpb >> >(d_A, d_b, d_IA, d_JA, d_X, d_preX, N);
		}
		hipMemcpy(X, d_X, sizeof(double)*phi.inner.size(), hipMemcpyDeviceToHost);

		if (step%check_step == 0){
			error = VectorMath<double>::rootOfSquareSum(phi.inner, pre.inner);
			cout << step << endl;
		}
		step++;
		/*
		transform(phi->inner.cbegin(), phi->inner.cend(), pre.inner.cbegin(),pre.inner.begin(), minus<T>());
		transform(pre.inner.begin(), pre.inner.end(), pre.inner.begin(), pre.inner.begin(), multiplies<T>());
		error = accumulate(pre.inner.begin(), pre.inner.end(), (T)0);//have to force convert to T type unless the result is always 0
		//error = inner_product(pre.inner.cbegin(), pre.inner.cend(), pre.inner.cbegin(),
		//	1, plus<T>(), plus<T>());
		error = sqrt(error);
		*/
		
	} while (error>converge_threhold&&step<max_step);

	//phi.inner . assign(X, X + N);
	
}



//vecotr data structure

__global__ void vector_kernel(CSR eq,CellField phi, CellField pre_phi,Mesh mesh){
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	double *A = eq.A.data();
	double *b = eq.b.data();
	int* IA = mesh.IA.data();
	int* JA = mesh.JA.data();
	double* X = phi.inner.data();
	double*preX = pre_phi.inner.data();

	if (tid < mesh.cellNum){
		double temp = 0;
		X[tid];
		/*
		for (int i = IA[tid] + 1; i < IA[tid + 1]; i++){
			temp += A[i] * preX[JA[i]];
		}
		X[tid] = (b[tid] - temp) / A[IA[tid]];
		*/
	}
}


void GPU_Jacobi_Solver_vector::solve(CSR eq, CellField& phi, Mesh mesh){

	
	CSR d_eq;
	CellField d_phi,d_pre_phi;
	Mesh d_mesh;

	
	/*
	double *A =new double[eq.A.size()];
	for (int i = 0; i < eq.A.size(); i++){
	A[i] = eq.A[i];
	}
	*/
	double *A = eq.A.data();
	double *b = eq.b.data();
	int *IA = mesh.IA.data();
	int  *JA = mesh.JA.data();
	double* X = phi.inner.data();
	int N = mesh.cellNum;
	double*d_A, *d_b, *d_X, *d_preX;
	int*d_IA, *d_JA;



	hipMalloc(&d_X, sizeof(double)*phi.inner.size());
	hipMalloc(&d_preX, sizeof(double)*phi.inner.size());
	hipMalloc(&d_A, sizeof(double)*eq.A.size());
	hipMalloc(&d_b, sizeof(double)*eq.b.size());
	hipMalloc(&d_IA, sizeof(int)*mesh.IA.size());
	hipMalloc(&d_JA, sizeof(int)*mesh.JA.size());

	hipMemcpy(d_A, A, sizeof(double)*eq.A.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(double)*eq.b.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_X, X, sizeof(double)*phi.inner.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_preX, X, sizeof(double)*phi.inner.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_IA, IA, sizeof(int)*mesh.IA.size(), hipMemcpyHostToDevice);
	hipMemcpy(d_JA, JA, sizeof(int)*mesh.JA.size(), hipMemcpyHostToDevice);

	d_eq.A.assign(d_A, d_A + eq.A.size() - 1);
	d_eq.b.assign(d_b, d_b + eq.b.size() - 1);
	d_mesh.IA.assign(d_IA, d_IA + mesh.IA.size() - 1);
	d_mesh.JA.assign(d_JA, d_JA + mesh.JA.size() - 1);
	d_phi.inner.assign(d_X, d_X + phi.inner.size() - 1);
	d_pre_phi.inner.assign(d_preX, d_preX + phi.inner.size() - 1);

	int tpb = 256;
	int bpg = (N + 256 - 1) / tpb;

	double error = 0; int step = 0;
	do{
		CellField pre = phi;

		for (int it = 0; it < check_step; it++){
			hipMemcpy(d_preX, d_X, sizeof(double)*phi.inner.size(), hipMemcpyDeviceToDevice);
			vector_kernel << <bpg, tpb >> >(d_eq,d_phi,d_pre_phi,d_mesh);
		}
		hipMemcpy(X, d_X, sizeof(double)*phi.inner.size(), hipMemcpyDeviceToHost);

		if (step%check_step == 0){
			error = VectorMath<double>::rootOfSquareSum(phi.inner, pre.inner);
			cout << step << endl;
		}
		step++;
		/*
		transform(phi->inner.cbegin(), phi->inner.cend(), pre.inner.cbegin(),pre.inner.begin(), minus<T>());
		transform(pre.inner.begin(), pre.inner.end(), pre.inner.begin(), pre.inner.begin(), multiplies<T>());
		error = accumulate(pre.inner.begin(), pre.inner.end(), (T)0);//have to force convert to T type unless the result is always 0
		//error = inner_product(pre.inner.cbegin(), pre.inner.cend(), pre.inner.cbegin(),
		//	1, plus<T>(), plus<T>());
		error = sqrt(error);
		*/

	} while (error>converge_threhold&&step<max_step);

	//phi.inner . assign(X, X + N);

}